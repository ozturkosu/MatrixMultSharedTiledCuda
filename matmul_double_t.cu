#include "hip/hip_runtime.h"
/* objective
 * 	C = A*B  // A[m][k], B[k][n], C[m][n]
 * compile: nvcc --gpu-architecture=compute_60 --gpu-code=sm_60 -O3 matmul_double_t.cu -o matmul_double_t
 */

#include <iostream>
#include <cstdlib>

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

#define TILE_WIDTH 16

#define EC(ans) { chkerr((ans), __FILE__, __LINE__); }
inline void chkerr(hipError_t code, const char *file, int line)
{
    if (code != hipSuccess)
    {
        std::cerr << "ERROR!!!:" << hipGetErrorString(code) << " File: " << file << " Line: " << line << '\n';
        exit(-1);
    }
}

void init (double *A, double *B, int M , int N, int K)
{
    for (int i = 0; i < M; ++i)
    {
        for (int j = 0; j < K; ++j)
        {
            A[i * K + j] = i * K + j;
        }
    }

    for (int i = 0; i < K; ++i)
    {
        for (int j = 0; j < N; ++j)
        {
            B[i * N + j] = i * N + j + 1;
        }
    }

}


void matmul_transpose_double_host(double* A, double* B, double* C, int M, int N, int K)
{
    for (int i = 0; i < M; ++i)
    {
        for (int j = 0; j < N; ++j)
        {
            double tmp = 0;

            for (int k = 0; k < K; ++k)
            {
                //tmp += A[i * K + k] * B[k * N + j];
                tmp +=A[i* K +k ] * B[j*N + k];
            }

            C[i * N + j] = tmp;
        }
    }
}

__global__ void matmul_double(double* A, double* B , double* B_t, double* C, int M, int N, int K)
{
    /// complete code
    // B matrix is N*K so we should find transpose of B


    //__syncthreads() ;

    transposeNoBankConflicts(B_t , B , K , N) ;

    //__syncthreads() ;


    int bx = blockIdx.x ;
    int by = blockIdx.y ;

    int tx = threadIdx.x ;
    int ty = threadIdx.y ;

    int row = by * TILE_WIDTH + ty ;
    int col = bx * TILE_WIDTH + tx ;

    __shared__ double SA[TILE_WIDTH][TILE_WIDTH] ;
    __shared__ double SB[TILE_WIDTH][TILE_WIDTH] ;

    double Csub = 0;

    int rowB = N;
    int colB = K;

    for (int i = 0; i < (K-1)/TILE_WIDTH +1 ; ++i)
    {
        /* code */
        //SA[ty][tx] = A[row*n + i * TILE_WIDTH + tx] ;
        //SB[ty][tx] = B[(i * TILE_WIDTH + ty )*n + col   ] ;

        if ( (row < M) && (i * TILE_WIDTH + tx < K ) ){
            SA[ty][tx] = A[row*N + i * TILE_WIDTH + tx] ;
        }
        else{
            SA[ty][tx] = 0;
        }

        if ( (col < colB ) && ( i * TILE_WIDTH + ty < rowB) ){
            SB[ty][tx] = B[(i*TILE_WIDTH + ty)*rowB + col] ;
            //SB[ty][tx] = B_t[(i*TILE_WIDTH + ty)*K + col] ;

        }
        else{
            SB[ty][tx] = 0;

        }



        __syncthreads() ;

        for (int k = 0; k < TILE_WIDTH; ++k){   
            Csub += SA[ty][k]*SB[k][tx] ;
        }

        __syncthreads() ;
        

    }

    //C[row*n + col] = Csub ;

    if ( (row < M ) && ( col < N )){
        C[ row * N + col] = Csub ;
    }



}

__device__ void transposeNoBankConflicts(float *odata, float *idata, int width, int height)
{
  __shared__ float tile[TILE_WIDTH][TILE_WIDTH+1];

  int xIndex = blockIdx.x * TILE_WIDTH + threadIdx.x;
  int yIndex = blockIdx.y * TILE_WIDTH + threadIdx.y;  
  int index_in = xIndex + (yIndex)*width;

  xIndex = blockIdx.y * TILE_WIDTH + threadIdx.x;
  yIndex = blockIdx.x * TILE_WIDTH + threadIdx.y;
  int index_out = xIndex + (yIndex)*height;

  for (int r=0; r < nreps; r++) 1{
    for (int i=0; i<TILE_WIDTH; i+=BLOCK_ROWS) {
      tile[threadIdx.y+i][threadIdx.x] = idata[index_in+i*width];
    }
  
    __syncthreads();
  
    for (int i=0; i<TILE_WIDTH; i+=BLOCK_ROWS) {
      odata[index_out+i*height] = tile[threadIdx.x][threadIdx.y+i];
    }
  }
}

void validate (double *host, double *gpu, int M, int N)
{
    for (int i = 0; i < M; ++i)
    {
        for (int j = 0; j < N; ++j)
        {
            if(std::abs(host[i * N + j] - gpu[i * N + j]) > 1e-3)
            {
                std::cerr << "possible error at position " << i << ',' << j << " host: " << host[i * N + j] << " device " << gpu[i * N + j] << '\n';
            }

        }
    }
}


int main(int argc, char *argv[])
{
    if(argc < 3)
    {
        std::cerr << "Usage: ./matmul_double M N K\n";
        exit(-1);
    }

    int M = std::atoi(argv[1]);
    int N = std::atoi(argv[2]);
    int K = std::atoi(argv[3]);

    /* Host alloc */
    double *hA = (double*) malloc (M * K * sizeof(double));
    double *hB = (double*) malloc (K * N * sizeof(double));
    double *hC = (double*) malloc (M * N * sizeof(double));
    double *dtohC = (double*) malloc (M * N * sizeof(double));

    /* Device alloc */
    /// complete code

    double *dA;
    double *dB;
    double *dC;
    double *dB_t;

    hipMalloc((void**) &dA, M*K * sizeof(double)) ;
    hipMalloc((void**) &dB, K*N * sizeof(double)) ;
    hipMalloc((void**) &dC, M*N * sizeof(double)) ;
    hipMalloc((void**) &dB_t, K*N * sizeof(double)) ;


    /* Initialize host memory*/
    init(hA, hB, M, N, K);

    /* host compute */
    matmul_double_host(hA, hB, hC, M, N, K);


    /* Copy from host to device */
    /// complete code
    hipMemcpy(dA,hA ,M*K * sizeof(double) , hipMemcpyHostToDevice ) ;
    hipMemcpy(dB,hB ,K*N * sizeof(double) , hipMemcpyHostToDevice ) ;

    
    /* call gpu kernel */
    /// complete code

    //Initialize the grid and block dimensions here
    dim3 dimGrid( (N - 1) / TILE_WIDTH + 1 , (M - 1)/ TILE_WIDTH + 1 , 1) ;
    dim3 dimBlock(TILE_WIDTH , TILE_WIDTH , 1) ;


    matmul_double<<<dimGrid, dimBlock>>>(dA, dB , dB_t, dC , M , N , K) ;


    /* Copy from device to host (dC -> dtohC) */
    /// complete code

    hipMemcpy(hC, dC , sizeof(double)*M*N , hipMemcpyDeviceToHost) ;

    /* host vs device validation */
    validate(hC, dtohC, M, N);


    /* be clean */
    free(hA);
    free(hB);
    free(hC);
    free(dtohC);

    /// add code to free gpu memory

    hipFree(dA) ;
    hipFree(dB) ;
    hipFree(dC) ;


    return 0;
}




