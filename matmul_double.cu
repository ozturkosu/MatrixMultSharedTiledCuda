/* objective
 * 	C = A*B  // A[m][k], B[k][n], C[m][n]
 * compile: nvcc --gpu-architecture=compute_60 --gpu-code=sm_60 -O3 matmul_double.cu -o matmul_double
  Using nvprof for this lab

            nvprof -- query-metrics
            nvprof dram_read_transactions ./test 1024 1024  128
            nvprof ./test 1024 1024 128 

                second line of result shows time for GPU kernel

                GFlop   ( 2MNK * 10^-9 ) / time (second)

 */

#include <iostream>
#include <cstdlib>

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

#define TILE_WIDTH 16

#define EC(ans) { chkerr((ans), __FILE__, __LINE__); }
inline void chkerr(hipError_t code, const char *file, int line)
{
    if (code != hipSuccess)
    {
        std::cerr << "ERROR!!!:" << hipGetErrorString(code) << " File: " << file << " Line: " << line << '\n';
        exit(-1);
    }
}

void init (double *A, double *B, int M , int N, int K)
{
    for (int i = 0; i < M; ++i)
    {
        for (int j = 0; j < K; ++j)
        {
            A[i * K + j] = i * K + j;
        }
    }

    for (int i = 0; i < K; ++i)
    {
        for (int j = 0; j < N; ++j)
        {
            B[i * N + j] = i * N + j + 1;
        }
    }

}


void matmul_double_host(double* A, double* B, double* C, int M, int N, int K)
{
    for (int i = 0; i < M; ++i)
    {
        for (int j = 0; j < N; ++j)
        {
            double tmp = 0;

            for (int k = 0; k < K; ++k)
            {
                tmp += A[i * K + k] * B[k * N + j];
            }

            C[i * N + j] = tmp;
        }
    }
}

__global__ void matmul_double(double* A, double* B , double* C, int M, int N, int K)
{
    /// complete code

    int bx = blockIdx.x ;
    int by = blockIdx.y ;

    int tx = threadIdx.x ;
    int ty = threadIdx.y ;

    int row = by * TILE_WIDTH + ty ;
    int col = bx * TILE_WIDTH + tx ;

    __shared__ double SA[TILE_WIDTH][TILE_WIDTH] ;
    __shared__ double SB[TILE_WIDTH][TILE_WIDTH] ;

    double Csub = 0;

    for (int i = 0; i < (K-1)/TILE_WIDTH +1 ; ++i)
    {
        /* code */
        //SA[ty][tx] = A[row*n + i * TILE_WIDTH + tx] ;
        //SB[ty][tx] = B[(i * TILE_WIDTH + ty )*n + col   ] ;

        if ( (row < M) && (i * TILE_WIDTH + tx < K ) ){
            SA[ty][tx] = A[row*K + i * TILE_WIDTH + tx] ;
        }
        else{
            SA[ty][tx] = 0;
        }

        if ( (col < N ) && ( i * TILE_WIDTH + ty < K) ){
            SB[ty][tx] = B[(i*TILE_WIDTH + ty)*N + col] ;
        }
        else{
            SB[ty][tx] = 0;
        }



        __syncthreads() ;

        for (int k = 0; k < TILE_WIDTH; ++k){   
            Csub += SA[ty][k] * SB[k][tx] ;
        }

        __syncthreads() ;
        

    }

    //C[row*n + col] = Csub ;

    if ( (row < M ) && ( col < N )){
        C[ row * N + col] = Csub ;
    }



}

void validate (double *host, double *gpu, int M, int N)
{
    for (int i = 0; i < M; ++i)
    {
        for (int j = 0; j < N; ++j)
        {
            if(std::abs(host[i * N + j] - gpu[i * N + j]) > 1e-3)
            {
                std::cerr << "possible error at position " << i << ',' << j << " host: " << host[i * N + j] << " device " << gpu[i * N + j] << '\n';
            }

        }
    }
}


int main(int argc, char *argv[])
{
    if(argc < 3)
    {
        std::cerr << "Usage: ./matmul_double M N K\n";
        exit(-1);
    }

    int M = std::atoi(argv[1]);
    int N = std::atoi(argv[2]);
    int K = std::atoi(argv[3]);

    /* Host alloc */
    double *hA = (double*) malloc (M * K * sizeof(double));
    double *hB = (double*) malloc (K * N * sizeof(double));
    double *hC = (double*) malloc (M * N * sizeof(double));
    double *dtohC = (double*) malloc (M * N * sizeof(double));

    /* Device alloc */
    /// complete code

    double *dA;
    double *dB;
    double *dC;

    hipMalloc((void**) &dA, M*K * sizeof(double)) ;
    hipMalloc((void**) &dB, K*N * sizeof(double)) ;
    hipMalloc((void**) &dC, M*N * sizeof(double)) ;


    /* Initialize host memory*/
    init(hA, hB, M, N, K);

    /* host compute */
    matmul_double_host(hA, hB, hC, M, N, K);


    /* Copy from host to device */
    /// complete code
    hipMemcpy(dA,hA ,M*K * sizeof(double) , hipMemcpyHostToDevice ) ;
    hipMemcpy(dB,hB ,K*N * sizeof(double) , hipMemcpyHostToDevice ) ;

    
    /* call gpu kernel */
    /// complete code

    //Initialize the grid and block dimensions here
    //dim3 dimGrid( ceil())
    dim3 dimGrid( (N - 1) / TILE_WIDTH + 1 , (M - 1)/ TILE_WIDTH + 1 , 1) ;
    //dim3 dimGrid( (M - 1) / TILE_WIDTH + 1 , (N - 1)/ TILE_WIDTH + 1 , 1) ;
    dim3 dimBlock(TILE_WIDTH , TILE_WIDTH , 1) ;


    matmul_double<<<dimGrid, dimBlock>>>(dA, dB , dC , M , N , K) ;


    /* Copy from device to host (dC -> dtohC) */
    /// complete code

    hipMemcpy(dtohC, dC , sizeof(double)*M*N , hipMemcpyDeviceToHost) ;

    /* host vs device validation */
    validate(hC, dtohC, M, N);


    /* be clean */
    free(hA);
    free(hB);
    free(hC);
    free(dtohC);

    /// add code to free gpu memory

    hipFree(dA) ;
    hipFree(dB) ;
    hipFree(dC) ;


    return 0;
}




